#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/tv_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void mask_kernel(const int count, const int size,
    const int H, const int W, Dtype* mask) {
  CUDA_KERNEL_LOOP(i, count) {
    const int unit_pos = i % size;
    if (unit_pos % W == W-1 || unit_pos / W == H-1) {
      mask[i] = (Dtype)0;
    } else {
      mask[i] = (Dtype)1;
    }
  }
}

template <typename Dtype>
void TVLossLayer<Dtype>::create_mask_gpu(const int count,
    const int H, const int W, Dtype* mask) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mask_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, H*W, H, W, mask);
}

template void TVLossLayer<float>::create_mask_gpu(const int count,
    const int H, const int W, float* mask);
template void TVLossLayer<double>::create_mask_gpu(const int count,
    const int H, const int W, double* mask);

template <typename Dtype>
void TVLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int W = bottom[0]->shape(-1);
  const int count = bottom[0]->count();
  caffe_gpu_sub(count-1, bottom[0]->gpu_data(), &bottom[0]->gpu_data()[1],
      x_diff_.mutable_gpu_data());
  caffe_gpu_mul(count, x_diff_.gpu_data(), mask_.gpu_data(),
      x_diff_.mutable_gpu_data());
  caffe_gpu_sub(count-W, bottom[0]->gpu_data(), &bottom[0]->gpu_data()[W],
      y_diff_.mutable_gpu_data());
  caffe_gpu_mul(count, y_diff_.gpu_data(), mask_.gpu_data(),
      y_diff_.mutable_gpu_data());
  caffe_gpu_mul(count, x_diff_.gpu_data(), x_diff_.gpu_data(),
      grad_norm_.mutable_gpu_data());  // X_diff^2
  caffe_gpu_mul(count, y_diff_.gpu_data(), y_diff_.gpu_data(),
      tmp_.mutable_gpu_data());  // Y_diff^2
  caffe_gpu_add(count, tmp_.gpu_data(), grad_norm_.gpu_data(),
      grad_norm_.mutable_gpu_data());  // X_diff^2 + Y_diff^2
  caffe_gpu_powx(count, grad_norm_.gpu_data(),
      (Dtype)this->layer_param_.tv_loss_param().beta()/2,
      tmp_.mutable_gpu_data());  // (X_diff^2 + Y_diff^2)^(beta/2)
  caffe_gpu_asum(count, tmp_.gpu_data(), top[0]->mutable_cpu_data());
}

template <typename Dtype>
void TVLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const int W = bottom[0]->shape(-1);
  const int count = bottom[0]->count();
  caffe_gpu_powx(count, grad_norm_.gpu_data(),
      (Dtype)this->layer_param_.tv_loss_param().beta() / 2 - 1,
      grad_norm_.mutable_gpu_data());
  caffe_gpu_scal(count, (Dtype)this->layer_param_.tv_loss_param().beta() / 2,
      grad_norm_.mutable_gpu_data());
  caffe_gpu_mul(count, x_diff_.gpu_data(), grad_norm_.gpu_data(),
      x_diff_.mutable_gpu_data());
  caffe_gpu_scal(count, (Dtype)2, x_diff_.mutable_gpu_data());  // dX_diff
  caffe_gpu_mul(count, y_diff_.gpu_data(), grad_norm_.gpu_data(),
      y_diff_.mutable_gpu_data());
  caffe_gpu_scal(count, (Dtype)2, y_diff_.mutable_gpu_data());  // dY_diff
  caffe_gpu_axpy(count, (Dtype)1, x_diff_.gpu_data(),
      bottom[0]->mutable_gpu_diff());
  caffe_gpu_axpy(count, (Dtype)1, y_diff_.gpu_data(),
      bottom[0]->mutable_gpu_diff());
  caffe_gpu_axpy(count-1, (Dtype)-1, x_diff_.gpu_data(),
      &bottom[0]->mutable_gpu_diff()[1]);
  caffe_gpu_axpy(count-W, (Dtype)-1, y_diff_.gpu_data(),
      &bottom[0]->mutable_gpu_diff()[W]);
  caffe_gpu_scal(count, top[0]->cpu_diff()[0], bottom[0]->mutable_gpu_diff());
}


INSTANTIATE_LAYER_GPU_FUNCS(TVLossLayer);

}  // namespace caffe
